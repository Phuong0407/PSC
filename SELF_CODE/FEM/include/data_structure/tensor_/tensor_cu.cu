#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "tensor_cu.hpp"

namespace data_structure {

// ✅ Custom atomicAdd for double (for GPUs < sm_60)
__device__ double atomicAdd_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

// ✅ CUDA kernel for dot product computation
__global__ void dot_kernel(const double* a, const double* b, double* result, std::size_t n) {
    __shared__ double temp[1024];  // Shared memory for block reduction
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int local_idx = threadIdx.x;

    temp[local_idx] = (idx < n) ? a[idx] * b[idx] : 0.0;
    __syncthreads();

    // Parallel reduction within block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (local_idx < stride) {
            temp[local_idx] += temp[local_idx + stride];
        }
        __syncthreads();
    }

    // Store result of block reduction
    if (local_idx == 0) {
        atomicAdd_double(result, temp[0]);  // ✅ Use custom atomicAdd for double
    }
}

// ✅ Dot product function for CUDA tensors
template<unsigned int Rank, unsigned int Dim, typename real_t>
std::unique_ptr<tensor_cu<Rank + Rank - 2, Dim, real_t>> dot(
    const tensor_cu<Rank, Dim, real_t>& a,
    const tensor_cu<Rank, Dim, real_t>& b)
{
    if (a.data_size() != b.data_size()) {
        throw std::invalid_argument("Size mismatch.");
    }

    std::size_t size = a.data_size();
    real_t* device_result;
    real_t host_result = 0.0;
    
    hipMalloc(&device_result, sizeof(real_t));
    hipMemcpy(device_result, &host_result, sizeof(real_t), hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int threads_per_block = 1024;
    int num_blocks = (size + threads_per_block - 1) / threads_per_block;
    dot_kernel<<<num_blocks, threads_per_block>>>(a.device_data_, b.device_data_, device_result, size);

    // Copy result back to host
    hipMemcpy(&host_result, device_result, sizeof(real_t), hipMemcpyDeviceToHost);
    hipFree(device_result);

    // ✅ Return result as tensor
    auto result_tensor = std::make_unique<tensor_cu<Rank + Rank - 2, Dim, real_t>>();
    result_tensor->init({host_result});
    return result_tensor;
}

// ✅ CUDA kernel for outer product computation
__global__ void outer_kernel(const double* a, const double* b, double* c, std::size_t m, std::size_t n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        c[row * n + col] = a[row] * b[col];
    }
}

// ✅ Outer product function for CUDA tensors
template<unsigned int Rank1, unsigned int Dim1, unsigned int Rank2, unsigned int Dim2, typename real_t>
std::unique_ptr<tensor_cu<Rank1 + Rank2, Dim1, real_t>> outer(
    const tensor_cu<Rank1, Dim1, real_t>& a,
    const tensor_cu<Rank2, Dim2, real_t>& b)
{
    std::size_t m = a.data_size();  // Number of rows
    std::size_t n = b.data_size();  // Number of columns
    std::size_t size = m * n;       // Total elements in the outer product result

    // Allocate memory for result tensor on GPU
    real_t* device_c;
    hipMalloc(&device_c, size * sizeof(real_t));

    // ✅ Define optimized grid and block dimensions
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((n + threads_per_block.x - 1) / threads_per_block.x,
                    (m + threads_per_block.y - 1) / threads_per_block.y);

    // ✅ Launch CUDA kernel
    outer_kernel<<<num_blocks, threads_per_block>>>(a.device_data_, b.device_data_, device_c, m, n);

    // ✅ Create and initialize result tensor
    auto result_tensor = std::make_unique<tensor_cu<Rank1 + Rank2, Dim1, real_t>>();
    hipMemcpy(result_tensor->device_data_, device_c, size * sizeof(real_t), hipMemcpyDeviceToDevice);

    hipFree(device_c);  // ✅ Free temporary GPU memory

    return result_tensor;
}

} // namespace data_structure
